#include "hip/hip_runtime.h"
#include "thrust/device_vector.h"
#include <thrust/logical.h>
#include <thrust/host_vector.h>
#include <thrust/execution_policy.h>
#include <thrust/for_each.h>
#include "thrust/sequence.h"
#include "thrust/device_new.h"
#include "ranges"

constexpr bool DEBUG = false;

struct label_if_res_functor {
    __host__ __device__
    // get the lowest label from residuals that is >0, otherwise inf to rule it out
    thrust::tuple<int, int> operator()(const thrust::tuple<int, int, int> &t) const {
        if (thrust::get<1>(t) > 0)
            return thrust::make_tuple(thrust::get<0>(t), thrust::get<2>(t));
        return thrust::make_tuple(-1, INT_MAX);
    }
};

struct excess_remains_functor {
    __host__ __device__
    bool operator()(const int node_excess) const {
        return node_excess > 0;
    }
};

struct pair_argmin_functor {
    __host__ __device__
    thrust::tuple<int, int> operator()(thrust::tuple<int, int> a, thrust::tuple<int, int> b) const {
       if (thrust::get<1>(a) < thrust::get<1>(b)) {
            return a;
        }
        return b;
    }
};

struct do_everything_functor {
    int *residuals; // residuals (N,N), rest N
    int *node_excess;
    int *node_labels;
    int N;

    __device__
    void operator()(const int node_id) const {
        const thrust::counting_iterator<int> first(0);
        const thrust::counting_iterator<int> last(N);

        const auto begin = make_zip_iterator(make_tuple(first, residuals + N * node_id, node_labels));
        const auto end = make_zip_iterator(make_tuple(last, residuals + N * (node_id + 1), node_labels + N));

        auto min_label_tuple = thrust::transform_reduce(thrust::device,
                                                  begin, end, label_if_res_functor(), thrust::make_tuple(-1, INT_MAX),
                                                  pair_argmin_functor());
        const auto min_label_idx = thrust::get<0>(min_label_tuple);
        const auto min_label_value = thrust::get<1>(min_label_tuple);

        if (DEBUG)
            printf("Node %d found min. label: %d, idx: %d\n", node_id, min_label_value, min_label_idx);

        if (min_label_value == INT_MAX) return;

        if (node_labels[node_id] > min_label_value) {
            const auto diff = thrust::min(node_excess[node_id], residuals[N * node_id + min_label_idx]);
            atomicSub(residuals + N * node_id + min_label_idx, diff);
            atomicAdd(residuals + N * min_label_idx + node_id, diff);
            if (DEBUG)
            printf("Pushing from %d to %d with diff %d\n", node_id, min_label_idx, diff);

            atomicSub(node_excess + node_id, diff);
            atomicAdd(node_excess + min_label_idx, diff);
        } else {
            if (DEBUG)
            printf("Elevate on %d\n", node_id);
            node_labels[node_id] = min_label_value + 1;
        }
    }
};

int main() {
    // adjacency matrix of initial capacities
    const std::vector<std::vector<int> > adj_c = {
        {0, 7, 0, 0},
        {0, 0, 6, 0},
        {0, 0, 0, 8},
        {9, 0, 0, 0}
    };

    const int N = adj_c.size();
    auto residuals = adj_c;

    auto node_levels = std::vector<int>(adj_c.size());
    node_levels[0] = static_cast<int>(node_levels.size());

    // f=c -> r=c-f=0
    std::ranges::fill(residuals[0], 0);
    for (int i = 0; i < residuals.size(); ++i) {
        residuals[i][0] = adj_c[0][i];
    }

    auto node_excess = std::vector<int>(adj_c.size());
    std::ranges::copy(adj_c[0], node_excess.begin());
    // flatten residuals using ranges view join
    auto residuals_flat = std::ranges::join_view(residuals);

    auto residuals_cuda = thrust::device_vector<int>(residuals_flat.begin(), residuals_flat.end());
    auto node_levels_cuda = thrust::device_vector<int>(node_levels.begin(), node_levels.end());
    auto node_excess_cuda = thrust::device_vector<int>(node_excess.begin(), node_excess.end());

    // todo: figure out fancier way
    auto node_indices_cuda = thrust::device_vector<int>(node_levels_cuda.size());
    sequence(node_indices_cuda.begin(), node_indices_cuda.end());

    bool excess_remains = true;
    const auto residuals_ptr = raw_pointer_cast(residuals_cuda.data());
    const auto node_excess_ptr = raw_pointer_cast(node_excess_cuda.data());
    const auto node_levels_ptr = raw_pointer_cast(node_levels_cuda.data());

    while (excess_remains) {
        for_each(thrust::device, node_indices_cuda.begin() + 1,  node_indices_cuda.end() - 1,
            do_everything_functor{
            residuals_ptr,
            node_excess_ptr,
            node_levels_ptr,
            N
        });

        excess_remains = any_of(thrust::device, node_excess_cuda.begin() + 1, node_excess_cuda.end() - 1, excess_remains_functor());
    }

    int result = -1;
    copy(node_excess_cuda.end() - 1, node_excess_cuda.end(), &result);
    printf("Result: %d\n", result);

    return 0;
}
